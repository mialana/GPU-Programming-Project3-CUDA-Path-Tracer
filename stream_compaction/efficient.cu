#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction
{
namespace Efficient
{
using StreamCompaction::Common::PerformanceTimer;

PerformanceTimer& timer()
{
    static PerformanceTimer timer;
    return timer;
}

__global__ void kernel_efficientUpSweep(const int n, const int iter, int* scan)
{
    int iterTarget = 1 << (iter + 1);
    int iterFactor = 1 << iter;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    index *= iterTarget;

    if (index + iterTarget - 1 < n)
    {
        scan[index + iterTarget - 1] += scan[index + iterFactor - 1];
    }
}

__global__ void kernel_efficientDownSweep(const int n, const int iter, int* scan)
{
    int iterTarget = 1 << (iter + 1);
    int iterFactor = 1 << iter;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    index = index * iterTarget;

    if (index + iterTarget - 1 < n)
    {
        int leftChild = scan[index + iterFactor - 1];
        scan[index + iterFactor - 1] = scan[index + iterTarget - 1];
        scan[index + iterTarget - 1] += leftChild;
    }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata)
{
    int numLayers = ilog2ceil(n);
    int paddedN = 1 << ilog2ceil(n);

    // create two device arrays
    int* dev_scan;

    hipMalloc((void**)&dev_scan, sizeof(int) * paddedN);
    checkCUDAError("CUDA malloc for scan array failed.");

    hipMemcpy(dev_scan, idata, sizeof(int) * paddedN, hipMemcpyHostToDevice);
    checkCUDAError("Memory copy from input data to scan array failed.");

    hipDeviceSynchronize();

    bool usingTimer = false;
    if (!timer().gpu_timer_started)  // added in order to call `scan` from other functions.
    {
        timer().startGpuTimer();
        usingTimer = true;
    }

    int blocks;

    for (int i = 0; i <= numLayers - 1; i++)
    {
        blocks = divup(paddedN / (1 << (i + 1)), BLOCK_SIZE);
        kernel_efficientUpSweep<<<blocks, BLOCK_SIZE>>>(paddedN, i, dev_scan);
        checkCUDAError("Perform Work-Efficient Scan Up Sweep Iteration CUDA kernel failed.");
    }

    Common::kernel_setDeviceArrayValue<<<1, 1>>>(dev_scan, paddedN - 1, 0);

    for (int i = numLayers - 1; i >= 0; i--)
    {
        blocks = divup(paddedN / (1 << (i + 1)), BLOCK_SIZE);
        kernel_efficientDownSweep<<<blocks, BLOCK_SIZE>>>(paddedN, i, dev_scan);
        checkCUDAError("Perform Work-Efficient Scan Down Sweep Iteration CUDA kernel failed.");
    }

    if (usingTimer)
    {
        timer().endGpuTimer();
    }

    hipMemcpy(odata, dev_scan, sizeof(int) * n, hipMemcpyDeviceToHost);

    hipFree(dev_scan);  // can't forget memory leaks!
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int* odata, const int* idata)
{
    // create device arrays
    int* dev_idata;
    int* dev_odata;

    int* dev_bools;
    int* dev_indices;

    hipMalloc((void**)&dev_idata, sizeof(int) * n);
    checkCUDAError("CUDA malloc for idata array failed.");

    hipMalloc((void**)&dev_odata, sizeof(int) * n);
    checkCUDAError("CUDA malloc for odata array failed.");

    hipMalloc((void**)&dev_bools, sizeof(int) * n);
    checkCUDAError("CUDA malloc for bools array failed.");

    hipMalloc((void**)&dev_indices, sizeof(int) * n);
    checkCUDAError("CUDA malloc for indices array failed.");

    hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
    checkCUDAError("Memory copy from input data to idata array failed.");
    hipMemcpy(dev_bools, odata, sizeof(int) * n, hipMemcpyHostToDevice);
    checkCUDAError("Memory copy from output data to odata array failed.");

    hipDeviceSynchronize();

    int* indices = new int[n]; // create cpu side indices array
    int* bools = new int[n];

    timer().startGpuTimer();

    int blocks = divup(n, BLOCK_SIZE);

    // reuse dev_idata for bools
    Common::kernMapToBoolean<<<blocks, BLOCK_SIZE>>>(n, dev_bools, dev_idata);

    hipMemcpy(bools, dev_bools, sizeof(int) * n, hipMemcpyDeviceToHost);
    checkCUDAError("Memory copy from device bools to indices array failed.");

    scan(n, indices, bools);

    hipMemcpy(dev_indices, indices, sizeof(int) * n, hipMemcpyHostToDevice);
    checkCUDAError("Memory copy from indices to device indices array failed.");

    Common::kernScatter<<<blocks, BLOCK_SIZE>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);

    timer().endGpuTimer();

    hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);

    hipFree(dev_idata);
    hipFree(dev_odata);
    hipFree(dev_bools);
    hipFree(dev_indices);

    return indices[n-1] + bools[n-1];
}
}  // namespace Efficient
}  // namespace StreamCompaction

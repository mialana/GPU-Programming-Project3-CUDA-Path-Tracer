#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction
{
namespace Thrust
{
using StreamCompaction::Common::PerformanceTimer;

PerformanceTimer& timer()
{
    static PerformanceTimer timer;
    return timer;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int* odata, const int* idata)
{
    // Copy data from host to device
    thrust::host_vector<int> host_idata(idata, idata + n);  // thrust host vector
    thrust::device_vector<int> dev_idata = host_idata;      // built-in assignment conversion
    thrust::device_vector<int> dev_odata(n);                // for output

    timer().startGpuTimer();

    thrust::exclusive_scan(dev_idata.begin(), dev_idata.end(), dev_odata.begin());

    timer().endGpuTimer();

    // copy result back to host
    thrust::copy(dev_odata.begin(), dev_odata.end(), odata);
}

void radixSort(int n, int* o_data, const int* i_data)
{
    thrust::device_vector<int> d_copy(i_data, i_data + n);

    bool usingTimer = false;
    if (!timer().gpu_timer_started)
    {
        timer().startGpuTimer();
        usingTimer = true;
    }

    thrust::sort(d_copy.begin(), d_copy.end());

    if (usingTimer)
    {
        timer().endGpuTimer();
    }

    thrust::copy(d_copy.begin(), d_copy.end(), o_data);
}

void radixSortByKey(int* d_keys, int* d_vals, int N)
{
    thrust::device_ptr<int> keys(d_keys);
    thrust::device_ptr<int> vals(d_vals);

    thrust::sort_by_key(keys, keys + N, vals);
}
}  // namespace Thrust
}  // namespace StreamCompaction
